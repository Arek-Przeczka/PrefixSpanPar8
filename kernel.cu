#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <math.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#define CHECK_BIT(var,pos) ((var) & (1<<(pos)))
#define BIT_SET(var,pos) ((var) |= (1ULL<<(pos)))
#define CALC_OFFSET (var,pos) (__popc((var) & (0xffffffff>>(32-pos))))

__device__ __constant__ int d_min_sup;
__device__ __constant__ int d_unique_item_count;

//function for tracking time of various operations 
void trackTime(int operation_type) {
	static std::chrono::time_point<std::chrono::steady_clock> start;
	static std::chrono::time_point<std::chrono::steady_clock> stop;
	static std::chrono::microseconds duration[7];
	static std::ofstream timefile("time.txt");
	stop = std::chrono::high_resolution_clock::now();
	switch (operation_type) {
		//load data from disc to RAM
		case 0:
			duration[0] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//transfer data from RAM to VRAM and memory allocation
		case 1:
			duration[1] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//scan database
		case 2:
			duration[2] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//prefix sum
		case 3:
			duration[3] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//stream compaction
		case 4:
			duration[4] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//transfer data from VRAM to RAM
		case 5:
			duration[5] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//save data on disc
		case 6:
			duration[6] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//other GPU operations
		case 7:
			duration[7] += std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
			break;
		//write time measurements to file
		case 8:
			if (timefile.is_open()) {
				for (int i = 0; i < 8/*sizeof(duration) / sizeof(duration[0])*/; i++) {
					timefile << duration[i].count() << "\n";
				}
				timefile.close();
			}
			else {
				std::cout << "Unable to open time file";
				exit(1);
			}
			break;
		//time tracking initialization - do nothing 
		case 9:
			break;

		default:
			std::cout << "Invalid value passed to trackTime function";
			exit(1);
			break;
	}
	start = std::chrono::high_resolution_clock::now();
}

inline void checkError(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
}

__global__ void firstScan(char* database, int* row_indexes_start, int* seq_val, int* new_row_idx_start, int* database_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; //database row
	int local_database_size = database_size[0];

	if (i < local_database_size) {
		unsigned int seq = 0xffffffff;
		unsigned int already_found_seq = 0x0;
		unsigned int seq_candidate;

		int j = row_indexes_start[i];
		char local_char = database[j];
		while (local_char != '.') {
			if (local_char != 44) {
				seq_candidate = CHECK_BIT(seq, local_char - 97);
				seq_candidate = already_found_seq | seq_candidate;
				if (seq_candidate != already_found_seq) {
					atomicAdd(&seq_val[local_char - 97], 1);
					new_row_idx_start[(local_char - 97) * local_database_size + i] = j + 1;
					already_found_seq = seq_candidate;
				}
			}
			j++;
			local_char = database[j];
		}
	}
}

__global__ void prepareVector(int* new_row_idx_start, int* target_index, int* seq_val, int* database_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; //database row
	int local_database_size = database_size[0];
	if (i < local_database_size) {
		for (int j = 0; j < d_unique_item_count; j++) { 
			target_index[j * local_database_size + i] = new_row_idx_start[j * local_database_size + i] > 0 && seq_val[j] >= d_min_sup ? 1 : 0;
		}
	}
}

__global__ void prepareVector2(int* new_row_idx_start, int* target_index, int* seq_val, int* database_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; //database row
	if (i < database_size[0] && new_row_idx_start[i] > 0) {
		target_index[i] = seq_val[target_index[i]] >= d_min_sup ? 1 : 0;
	}
}

__global__ void streamCompaction(int* new_row_idx_start, int* target_index, int* row_indexes_start, int* database_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < database_size[0]) {
		if (i == 0) {
			if (target_index[i] > 0) {
				row_indexes_start[target_index[i] - 1] = new_row_idx_start[i];
			}
		}
		else if (target_index[i] > target_index[i - 1]) {
			row_indexes_start[target_index[i] - 1] = new_row_idx_start[i];
		}
	}
}

__global__ void prepareSeq(int* seq_val, int* target_index, int* seq_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < seq_size[0]) {
		target_index[i] = seq_val[i] >= d_min_sup ? 1 : 0;
	}
}

__global__ void trimSeq(char* seq, int* seq_val, char* seq_trimmed, int* seq_val_trimmed, int* target_index, int* sup_seq_idx, int* old_sup_seq_idx, int* single_seq_size, int* seq_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < seq_size[0]) {
		if (i == 0) {
			if (target_index[i] > 0) {
				for (int j = 0; j < single_seq_size[0]; j++) {
					seq_trimmed[(target_index[i] - 1) * single_seq_size[0] + j] = seq[i * single_seq_size[0] + j];
				}
				seq_val_trimmed[target_index[i] - 1] = seq_val[i];
				old_sup_seq_idx[target_index[i] - 1] = sup_seq_idx[i];
			}
		}
		else if (target_index[i] > target_index[i - 1]) {
			for (int j = 0; j < single_seq_size[0]; j++) {
				seq_trimmed[(target_index[i] - 1) * single_seq_size[0] + j] = seq[i * single_seq_size[0] + j];
			}
			seq_val_trimmed[target_index[i] - 1] = seq_val[i];
			old_sup_seq_idx[target_index[i] - 1] = sup_seq_idx[i];
		}
	}
}

__global__ void firstGenerateNewSeq(char* seq, char* seq_trimmed, int* seq_val_trimmed, int* sup_seq_idx, int* single_seq_size, unsigned int* append,
	unsigned int* assemble, unsigned int* last_element, int* sup_database_size, int* seq_val_idx, int* new_row_idx_size, int* seq_val_trimmed_size) {

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int seq_idx = 0;

	//Determine where given thread should write their sequences:
	for (int j = 0; j < i; j++) {
		seq_idx += 2 * seq_val_trimmed_size[0] - 1 - j;
	}

	seq_val_idx[i] = seq_idx;

	//create new sequences:
	//create appends
	for (int j = 0; j < seq_val_trimmed_size[0]; j++) {
		seq[seq_idx * single_seq_size[0]] = seq_trimmed[i];
		seq[seq_idx * single_seq_size[0] + 1] = ',';
		seq[seq_idx * single_seq_size[0] + 2] = seq_trimmed[j];
		BIT_SET(append[i], seq_trimmed[j] - 97);
		sup_seq_idx[seq_idx] = i;
		seq_idx++;
	}

	//create assemblages
	for (int j = i + 1; j < seq_val_trimmed_size[0]; j++) {
		seq[seq_idx * single_seq_size[0]] = seq_trimmed[i];
		seq[seq_idx * single_seq_size[0] + 1] = '_';
		seq[seq_idx * single_seq_size[0] + 2] = seq_trimmed[j];
		BIT_SET(assemble[i], seq_trimmed[j] - 97);
		sup_seq_idx[seq_idx] = i;
		seq_idx++;
	}

	BIT_SET(last_element[i], seq_trimmed[i] - 97);
	sup_database_size[i] = seq_val_trimmed[i];
	new_row_idx_size[i + 1] = seq_val_trimmed[i] * (2 * seq_val_trimmed_size[0] - i - 1);

}

__global__ void scanDatabase(int* database_size, int* sub_database_size, unsigned int* append, unsigned int* assemble, int* row_indexes_start,
	char* database, int* seq_val, int* new_row_idx_start, int* seq_val_idx, unsigned int* last_element, int* new_database_start_idx, int* target_index, int* seq_val_trimmed_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < database_size[0]) {
		int local_sub_database_start;
		int local_sub_database_size;
		bool comma_found = false;
		int min_boundary = 0;
		int max_boundary = seq_val_trimmed_size[0];
		int middle_point;
		
		//determine which sub database given thread is searching
		//special case - first sub database:
		if (i < sub_database_size[0]) {
			middle_point = 0;
			local_sub_database_size = sub_database_size[middle_point];
			local_sub_database_start = 0;
		}
		//standard case, locate database using binary search:
		else {
			while (true) {
				middle_point = (min_boundary + max_boundary) / 2;
				if (i >= sub_database_size[middle_point]) {
					if (i < sub_database_size[middle_point + 1]) {
						local_sub_database_size = sub_database_size[middle_point + 1] - sub_database_size[middle_point];
						local_sub_database_start = sub_database_size[middle_point];
						middle_point++;
						break;
					}
					else {
						min_boundary = middle_point;
					}
				}
				else {
					max_boundary = middle_point;
				}
			}
		}

		//find appends
		unsigned int local_seq = append[middle_point];
		unsigned int already_found_seq = 0x0;
		unsigned int seq_candidate;
		int l = row_indexes_start[i];
		char local_char = database[l]; 
		int idx_offset;
		int append_idx_offset = __popc(local_seq);

		while (local_char != '.') {
			if (local_char == 44) {
				comma_found = true;
			}
			else {
				seq_candidate = CHECK_BIT(local_seq, local_char - 97);
				seq_candidate = already_found_seq | seq_candidate;
				if (seq_candidate != already_found_seq && comma_found == true) {
					idx_offset = __popc((local_seq) & (0xffffffff >> (32 - local_char + 97)));
					atomicAdd(&seq_val[seq_val_idx[middle_point] + idx_offset], 1);
					new_row_idx_start[idx_offset * local_sub_database_size + i - local_sub_database_start + new_database_start_idx[middle_point]] = l + 1;
					target_index[idx_offset * local_sub_database_size + i - local_sub_database_start + new_database_start_idx[middle_point]] = seq_val_idx[middle_point] + idx_offset;
					already_found_seq = seq_candidate;
				}
			}
			l++;
			local_char = database[l];
		}

		//find assemblages
		l = row_indexes_start[i];
		local_char = database[l];
		already_found_seq = 0x0;
		local_seq = assemble[middle_point];
		unsigned int local_prefix = last_element[middle_point];
		unsigned int found_prefix = local_prefix;

		while (local_char != '.') {
			if (local_char == 44) { //if comma is encountered prefix needs to be found again
				found_prefix = 0x0;
			}
			else {
				seq_candidate = CHECK_BIT(local_seq, local_char - 97);
				seq_candidate = already_found_seq | seq_candidate; 
				found_prefix = found_prefix | CHECK_BIT(local_prefix, local_char - 97);
				if (seq_candidate != already_found_seq && found_prefix == local_prefix) {
					idx_offset = __popc((local_seq) & (0xffffffff >> (32 - local_char + 97))) + append_idx_offset;
					atomicAdd(&seq_val[seq_val_idx[middle_point] + idx_offset], 1);
					new_row_idx_start[idx_offset * local_sub_database_size + i - local_sub_database_start + new_database_start_idx[middle_point]] = l + 1;
					target_index[idx_offset * local_sub_database_size + i - local_sub_database_start + new_database_start_idx[middle_point]] = seq_val_idx[middle_point] + idx_offset;
					already_found_seq = seq_candidate;
				}
			}
			l++;
			local_char = database[l];
		}
	}
}

__global__ void correctOldSupSeqIdx(int* old_sup_seq_idx, int* adj_diff) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//Correct old superior sequence index values so that difference between adjecent values is <= 1
	if (i == 0) {
		adj_diff[i] = old_sup_seq_idx[i];
	}
	else {
		adj_diff[i] = old_sup_seq_idx[i] - old_sup_seq_idx[i - 1] > 1 ? old_sup_seq_idx[i] - old_sup_seq_idx[i - 1] - 1 : 0;
	}
}

__global__ void calculateNewSeqPos(char* seq_trimmed, int* seq_start, int* assemble_start, int* old_sup_seq_idx, int* single_seq_size, int* seq_val_trimmed_size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//establish how many sequences have the same sup_seq_idx and where assemblages begin
	if (i < seq_val_trimmed_size[0]) {
		//special code for first sequence
		if (i == 0) {
			seq_start[i] = i;
			//check if this sequence is an assemble
			if (seq_trimmed[(i + 1) * (single_seq_size[0] - 2) - 2] == '_') {
				assemble_start[old_sup_seq_idx[i]] = i;
			}
			//check if there is only one sequence in database
			if (i == seq_val_trimmed_size[0] - 1) {
				seq_start[old_sup_seq_idx[i] + 1] = i + 1;
				if (seq_trimmed[(i + 1) * (single_seq_size[0] - 2) - 2] == ',') {
					assemble_start[old_sup_seq_idx[i]] = i + 1;
				}
			}
		}
		else {
			//special code for last sequence
			if (i == seq_val_trimmed_size[0] - 1) {
				seq_start[old_sup_seq_idx[i] + 1] = i + 1;
				//check if last sequence is an append
				if (seq_trimmed[i * (single_seq_size[0] - 2) - 2] == ',') {
					//if true then previous sub database does not have any assemblages, so set assemble start[i] = seq_start[i]
					assemble_start[old_sup_seq_idx[i]] = i + 1;
				}
				// check if this sequence is an append
				if (seq_trimmed[(i + 1) * (single_seq_size[0] - 2) - 2] == ',') {
					//if true then last sub database has only appends, so set assemble start[i] = seq_start[i + 1]
					assemble_start[old_sup_seq_idx[i]] = i + 1;
				}
			}

			//default code:
			//new sub database starts when superior sequence index changes
			if (old_sup_seq_idx[i] > old_sup_seq_idx[i - 1]) {
				seq_start[old_sup_seq_idx[i]] = i;
				// check if previous sequence was an append
				if (seq_trimmed[i * (single_seq_size[0] - 2) - 2] == ',') {
					//if true then previous sub database does not have any assemblages, so set assemble start[i-1] = seq_start[i]
					assemble_start[old_sup_seq_idx[i] - 1] = i;
				}
				// check if this sequence is an assemblage
				if (seq_trimmed[(i + 1) * (single_seq_size[0] - 2) - 2] == '_') {
					//if true then this sub database has only assemblages, so set assemble start[i] = seq_start[i]
					assemble_start[old_sup_seq_idx[i]] = i;
				}
			}
			//check where assemblages start
			if (seq_trimmed[(i + 1) * (single_seq_size[0] - 2) - 2] == '_' && seq_trimmed[i * (single_seq_size[0] - 2) - 2] == ',') {
				assemble_start[old_sup_seq_idx[i]] = i;
			}
		}
	}
}


__global__ void generateNewSeq(char* seq, char* seq_trimmed, int* seq_val_trimmed, int* sup_seq_idx, int* old_sup_seq_idx, int* single_seq_size, unsigned int* append,
	unsigned int* assemble, unsigned int* last_element, int* seq_start, int* assemble_start, int* seq_val_idx, int* sup_database_size, int* new_row_idx_size, int* seq_val_trimmed_size) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int seq_idx = 0;
	int n_ap;
	int n_as;

	if (i < seq_val_trimmed_size[0]) {
		//Determine where given thread should write their sequences:
		//seq_idx += all new sequences from sub databases that come before sub database in which "i" sequence is in
		for (int j = 0; j < old_sup_seq_idx[i]; j++) {
			n_ap = assemble_start[j] - seq_start[j];
			n_as = seq_start[j + 1] - assemble_start[j];
			seq_idx += n_ap * n_ap + n_ap * (n_ap - 1) / 2 + n_ap * n_as + n_as * (n_as - 1) / 2;;
		}

		n_ap = assemble_start[old_sup_seq_idx[i]] - seq_start[old_sup_seq_idx[i]];
		n_as = seq_start[old_sup_seq_idx[i] + 1] - assemble_start[old_sup_seq_idx[i]];

		//seq_idx += all new sequences from database in which "i" sequence is in generated from append that come before "i" sequence
		for (int j = seq_start[old_sup_seq_idx[i]]; j < i && j < assemble_start[old_sup_seq_idx[i]]; j++) {
			seq_idx += 2 * n_ap - j - 1 + seq_start[old_sup_seq_idx[i]];
		}

		//seq_idx += all new sequences from database in which "i" sequence is in generated from assemblages that come before "i" sequence (if any)
		for (int j = assemble_start[old_sup_seq_idx[i]]; j < i; j++) {
			seq_idx += n_ap + n_as - j - 1 + assemble_start[old_sup_seq_idx[i]];
		}

		seq_val_idx[i] = seq_idx;

		//create new sequences:
		//create appends
		for (int j = seq_start[old_sup_seq_idx[i]]; j < assemble_start[old_sup_seq_idx[i]]; j++) {	
			for (int k = 0; k < single_seq_size[0] - 2; k++) {
				seq[seq_idx * single_seq_size[0] + k] = seq_trimmed[i * (single_seq_size[0] - 2) + k];
			}
			seq[(seq_idx + 1) * single_seq_size[0] - 2] = ',';
			seq[(seq_idx + 1) * single_seq_size[0] - 1] = seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1];
			BIT_SET(append[i], seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1] - 97);
			sup_seq_idx[seq_idx] = i;
			seq_idx++;
		}

		//create assemblages
		if (i < assemble_start[old_sup_seq_idx[i]]) { 
			for (int j = i + 1; j < assemble_start[old_sup_seq_idx[i]]; j++) {
				for (int k = 0; k < single_seq_size[0] - 2; k++) {
					seq[seq_idx * single_seq_size[0] + k] = seq_trimmed[i * (single_seq_size[0] - 2) + k];
				}
				seq[(seq_idx + 1) * single_seq_size[0] - 2] = '_';
				seq[(seq_idx + 1) * single_seq_size[0] - 1] = seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1];
				BIT_SET(assemble[i], seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1] - 97);
				sup_seq_idx[seq_idx] = i;
				seq_idx++;
			}
			new_row_idx_size[i + 1] = seq_val_trimmed[i] * (2 * n_ap - i - 1 + seq_start[old_sup_seq_idx[i]]);//???
		}
		else {
			for (int j = i + 1; j < seq_start[old_sup_seq_idx[i] + 1]; j++) {
				for (int k = 0; k < single_seq_size[0] - 2; k++) {
					seq[seq_idx * single_seq_size[0] + k] = seq_trimmed[i * (single_seq_size[0] - 2) + k];
				}
				seq[(seq_idx + 1) * single_seq_size[0] - 2] = '_';
				seq[(seq_idx + 1) * single_seq_size[0] - 1] = seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1];
				BIT_SET(assemble[i], seq_trimmed[(j + 1) * (single_seq_size[0] - 2) - 1] - 97);
				sup_seq_idx[seq_idx] = i;
				seq_idx++;
			}
			new_row_idx_size[i + 1] = seq_val_trimmed[i] * (n_as + n_ap - i - 1 + assemble_start[old_sup_seq_idx[i]]);//???
		}

		//create last elements
		for (int j = single_seq_size[0] - 3; seq_trimmed[i * (single_seq_size[0] - 2) + j] != ',' && j > -1; j--) {
			if (seq_trimmed[i * (single_seq_size[0] - 2) + j] != '_') {
				BIT_SET(last_element[i], seq_trimmed[i * (single_seq_size[0] - 2) + j] - 97);
			}
		}
		
		sup_database_size[i] = seq_val_trimmed[i];
	}
}

int main(int argc, char** argv)
{
	auto start = std::chrono::high_resolution_clock::now();

	//enum for measuring exec time of different parts of program
	/*enum TimeMeasure
	{
		disk_to_RAM, RAM_to_VRAM, scan_db, prefix_sum, stream_compaction, VRAM_to_RAM, RAM_to_disk, other_GPU, write_results, init
	};
	trackTime(TimeMeasure::init);*/
	
	int h_unique_item_count; //value stating how many unique items are in database
	//check argc
	switch (argc) {
	case 4:
		h_unique_item_count = 26;
		break;
	case 5:
		h_unique_item_count = std::stoi(argv[4]);
		break;
	default:
		std::cout << "Incorrect number of input arguments\n";
		return 0;
	}

	float h_float_min_sup = std::stof(argv[3]); //minimum support

	if (h_float_min_sup > 1 || h_float_min_sup <= 0) {
		std::cout << "Incorrect minimum support value\n";
		return 0;
	}

	//host variables:
	std::ifstream file(argv[1]); //input database file
	std::ofstream output(argv[2]); //output sequences file
	std::string output_string; // string for writing to output file
	std::string h_database; //all database items in a single string
	std::vector<int> h_row_idx_start; //starting point of all rows in database
	std::vector<int> h_seq_val(h_unique_item_count); //frequency of sequences
	std::vector<char> h_seq(h_unique_item_count); //potential frequent sequences
	//std::vector<int> h_target_index; //DEBUG
	//std::vector<int> h_debug; //DEBUG
	std::vector<std::string> h_freq_seq; //found frequent sequences
	std::vector<int> h_seq_start; //index for first sequence in all sub databases, used to calculate number of appends and assemblages in sub databases
	std::vector<int> h_assemble_start;//index for first assemblage in all sub databases, used to calculate number of appends and assemblages in sub databases

	int h_database_size;//number of rows in database
	int h_new_database_size; //number of database rows in next iteration
	int h_seq_size;//number of potential frequent sequences
	int h_single_seq_size = 1;//single sequence size = 2n + 1, where n = number of algorithm iterations
	int h_trimmed_seq_size;//how many frequent sequences were found in given iteration
	int h_seq_start_size; 
	int h_min_sup; //minimum support expressed as number of rows in database
	

	//device variables:
	char* d_database;//all database items in a single string
	int* d_row_idx_start;//starting point of all rows in database
	int* d_seq_val;//frequency of sequences
	int* d_seq_val_trimmed;//frequency of sequences after trimming infrequent sequences
	int* d_new_row_idx_start;//starting point of all rows in database in next iteration
	int* d_database_size; //number of rows in database
	int* d_target_index; //variable for identifying which values of d_new_row_idx_start should be written to d_row_idx_start during stream compaction
	unsigned int* d_append; //appends that are being searched for in given datbase
	unsigned int* d_assemble; //assemblages thet are being searched for in given database
	unsigned int* d_last_element; //items that make up last element of prefix in given database
	int* d_sup_seq_idx; //index identifying superior database of given sequence
	int* d_old_sup_seq_idx; //index identifying superior database of given sequence from previous iteration
	int* d_sup_database_size; // number of rows that contained frequent sequences in previous iteration
	int* d_seq_val_idx; //index for identyfing which value in d_seq_val should be incremented after finding frequent sequence
	int* d_new_row_idx_size; //number of potential new database rows in next iteration
	int* d_adj_diff; //used to correct values in d_old_sup_seq_idx
	
	int* d_single_seq_size;//number of chars that form a single sequence = 2n + 1, where n = number of algorithm iterations
	char* d_seq;//potential frequent sequences
	char* d_seq_trimmed;//found frequent sequences
	int* d_seq_start;//index for first sequence in all sub databases, used to calculate number of appends and assemblages in sub databases
	int* d_assemble_start;//index for first assemblage in all sub databases, used to calculate number of appends and assemblages in sub databases
	int* d_seq_val_trimmed_size;//how many frequent sequences were found in given iteration
	int* d_new_database_size;//number of database rows in next iteration
	int* d_seq_size;

	//host variables for launching kernels:
	dim3 grid_size;
	dim3 block_size;
	const unsigned int max_block_size = 512;

	hipError_t cudaStatus; //error container
	/*size_t h_free_memory;
	size_t h_total_memory;
	unsigned long long h_requied_memory;
	unsigned long long h_allocation_size;
	int h_partition_num;
	bool h_alloc_success;*/


	for (int i = 0; i < h_unique_item_count; i++) {
		h_seq[i] = 97 + i;
	}

	//open file with input database
	if (file.is_open()) {
		std::string line;
		int indexes_iterator = 0;

		h_row_idx_start.push_back(0);

		//load database
		while (std::getline(file, line)) {
			h_database += line;
			indexes_iterator += line.size();
			h_row_idx_start.push_back(indexes_iterator);
		}
		h_row_idx_start.pop_back();
		file.close();
	}
	else {
		std::cout << "Unable to open file";
		return 0;
	}

	h_database_size = h_row_idx_start.size();
	h_new_database_size = h_database_size * h_unique_item_count;
	h_min_sup = ceil(h_database_size * h_float_min_sup);

	//trackTime(TimeMeasure::disk_to_RAM);

	//allocate memory on GPU and copy data
	cudaStatus = hipMalloc((void**)&d_database, sizeof(char) * h_database.size());
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_database, h_database.data(), sizeof(char) * h_database.size(), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_row_idx_start, sizeof(int) * h_row_idx_start.size());
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_row_idx_start, h_row_idx_start.data(), sizeof(int) * h_row_idx_start.size(), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_val, sizeof(int) * h_seq_val.size());
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_seq_val, 0, sizeof(int) * h_seq_val.size());
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq, sizeof(char) * h_seq.size());
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_seq, h_seq.data(), sizeof(char) * h_seq.size(), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_new_row_idx_start, sizeof(int) * h_unique_item_count * h_database_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_new_row_idx_start, 0, sizeof(int) * h_unique_item_count * h_database_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_database_size, sizeof(int));
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_database_size, &h_database_size, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_size, sizeof(int));
	checkError(cudaStatus);

	//trackTime(TimeMeasure::RAM_to_VRAM);

	//determine block and grid size
	if (h_database_size > max_block_size) {
		grid_size = { (unsigned int)(h_database_size + max_block_size - 1) / max_block_size, 1, 1 };
		block_size = { max_block_size, 1, 1 };
	}
	else {
		grid_size = { 1, 1, 1 };
		block_size = { (unsigned int)h_database_size, 1, 1 };
	}

	//launch first database scan to find all length 1 sequences
	firstScan <<<grid_size, block_size >>> (d_database, d_row_idx_start, d_seq_val, d_new_row_idx_start, d_database_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	//trackTime(TimeMeasure::scan_db);

	cudaStatus = hipMalloc((void**)&d_target_index, sizeof(int) * h_unique_item_count * h_database_size);
	checkError(cudaStatus);

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_min_sup), &h_min_sup, sizeof(int));
	checkError(cudaStatus);

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(d_unique_item_count), &h_unique_item_count, sizeof(int));
	checkError(cudaStatus);

	//trackTime(TimeMeasure::RAM_to_VRAM);

	//check which rows didn't contain frequent sequence
	prepareVector <<<grid_size, block_size >>> (d_new_row_idx_start, d_target_index, d_seq_val, d_database_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	//trackTime(TimeMeasure::other_GPU);

	//DEBUG
	/*h_target_index.resize(h_new_database_size);
	cudaStatus = hipMemcpy(h_target_index.data(), d_new_row_idx_start, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);*/
	//DEBUG_END

	//Prefix sum with thrust
	thrust::device_ptr<int> d_thrust_ptr = thrust::device_pointer_cast(d_target_index);
	thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_new_database_size, d_thrust_ptr);

	//trackTime(TimeMeasure::prefix_sum);

	cudaStatus = hipMemcpy(&h_database_size, d_target_index + (h_new_database_size - 1), sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	cudaStatus = hipMemcpy(d_database_size, &h_new_database_size, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	hipFree(d_row_idx_start);
	cudaStatus = hipMalloc((void**)&d_row_idx_start, sizeof(int) * h_database_size);
	checkError(cudaStatus);

	//determine block and grid size
	if (h_new_database_size > max_block_size) {
		grid_size = { (unsigned int)(h_new_database_size + max_block_size - 1) / max_block_size, 1, 1 };
		block_size = { max_block_size, 1, 1 };
	}
	else {
		grid_size = { 1, 1, 1 };
		block_size = { (unsigned int)h_new_database_size, 1, 1 };
	}

	//Stream compaction to leave only rows with frequent sequences
	streamCompaction <<<grid_size, block_size >>> (d_new_row_idx_start, d_target_index, d_row_idx_start, d_database_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	//trackTime(TimeMeasure::stream_compaction);

	cudaStatus = hipMemcpy(d_database_size, d_target_index + (h_new_database_size - 1), sizeof(int), hipMemcpyDeviceToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMemcpy(h_seq_val.data(), d_seq_val, sizeof(int) * h_seq_val.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	//trackTime(TimeMeasure::VRAM_to_RAM);
	//DEBUG
	/*h_target_index.resize(h_database_size);
	cudaStatus = hipMemcpy(h_target_index.data(), d_row_idx_start, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);*/
	//DEBUG_END

	//Free memory on device
	hipFree(d_new_row_idx_start);
	hipFree(d_target_index);

	//check which sequences to trim
	cudaStatus = hipMalloc((void**)&d_target_index, sizeof(int) * h_seq_val.size());
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_size, sizeof(int));
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_seq_size, &h_unique_item_count, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	//trackTime(TimeMeasure::RAM_to_VRAM);

	grid_size = { 1, 1, 1 };
	block_size = { (unsigned int)h_unique_item_count, 1, 1 };

	prepareSeq <<<grid_size, block_size >>> (d_seq_val, d_target_index, d_seq_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	//Prefix sum with thrust
	d_thrust_ptr = thrust::device_pointer_cast(d_target_index);
	thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_seq_val.size(), d_thrust_ptr);

	cudaStatus = hipMemcpy(&h_trimmed_seq_size, d_target_index + (h_seq_val.size() - 1), sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_val_trimmed, sizeof(int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_trimmed, sizeof(char) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_single_seq_size, sizeof(int));
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_single_seq_size, &h_single_seq_size, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_sup_seq_idx, sizeof(int) * h_unique_item_count);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_sup_seq_idx, 0, sizeof(int) * h_unique_item_count);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_old_sup_seq_idx, sizeof(int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	//trim infrequent sequences
	trimSeq <<<grid_size, block_size >>> (d_seq, d_seq_val, d_seq_trimmed, d_seq_val_trimmed, d_target_index, d_sup_seq_idx, d_old_sup_seq_idx, d_single_seq_size, d_seq_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	//trackTime(TimeMeasure::other_GPU);

	//copy results to CPU
	h_seq_val.resize(h_trimmed_seq_size);
	cudaStatus = hipMemcpy(h_seq_val.data(), d_seq_val_trimmed, sizeof(int) * h_seq_val.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	h_seq.resize(h_trimmed_seq_size);
	cudaStatus = hipMemcpy(h_seq.data(), d_seq_trimmed, sizeof(char) * h_seq.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	
	for (int i = 0; i < h_seq.size(); i++) {
		output_string += h_seq[i];
		output_string += ' ';
		output_string += std::to_string(h_seq_val[i]);
		output_string += '\n';
	}
	
	/*for (int i = 0; i < h_seq.size(); i++) {
		h_freq_seq.push_back({});
		h_freq_seq[i].push_back(h_seq[i]);
	}*/

	if (h_seq_val.size() == 0) {
		return 0;
	}

	//trackTime(TimeMeasure::VRAM_to_RAM);

	//print frequent sequences 
	/*for (int i = 0; i < std::size(h_seq_val); i++) {
		std::cout << h_freq_seq[i] << " " << h_seq_val[i] << "\n";
	}*/

	//write found sequences to output file
	if (output.is_open()) {
		output << output_string;
		/*for (int i = 0; i < std::size(h_seq_val); i++) {
			output << h_freq_seq[i] << " " << h_seq_val[i] << "\n";
		}*/
	}
	else {
		std::cout << "Unable to open output file";
		return 0;
	}

	//trackTime(TimeMeasure::RAM_to_disk);

	hipFree(d_seq);
	hipFree(d_seq_val);
	hipFree(d_target_index);
	hipFree(d_sup_seq_idx);

	//Prepare new sequences
	h_single_seq_size = h_single_seq_size + 2; //single sequence size = 2n + 1, where n = number of algorithm iterations
	h_seq_size = h_trimmed_seq_size * h_trimmed_seq_size + h_trimmed_seq_size * (h_trimmed_seq_size - 1) / 2;

	cudaStatus = hipMalloc((void**)&d_seq, sizeof(char) * h_seq_size * h_single_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_val, sizeof(int) * h_seq_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_seq_val, 0, sizeof(int) * h_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_sup_seq_idx, sizeof(int) * h_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMemcpy(d_single_seq_size, &h_single_seq_size, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_append, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_append, 0, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_assemble, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_assemble, 0, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_last_element, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_last_element, 0, sizeof(unsigned int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_seq_val_trimmed_size, sizeof(int));
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_seq_val_trimmed_size, &h_trimmed_seq_size, sizeof(int), hipMemcpyHostToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_sup_database_size, sizeof(int) * h_trimmed_seq_size);
	checkError(cudaStatus); 

	cudaStatus = hipMalloc((void**)&d_seq_val_idx, sizeof(int) * h_trimmed_seq_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_new_row_idx_size, sizeof(int) * (h_trimmed_seq_size + 1));
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_new_row_idx_size, 0, sizeof(int));
	checkError(cudaStatus);

	grid_size = { 1, 1, 1 };
	block_size = { (unsigned int)h_trimmed_seq_size, 1, 1 };

	firstGenerateNewSeq <<<grid_size, block_size >>> (d_seq, d_seq_trimmed, d_seq_val_trimmed, d_sup_seq_idx, d_single_seq_size, d_append, d_assemble, d_last_element, d_sup_database_size, d_seq_val_idx, d_new_row_idx_size, d_seq_val_trimmed_size);

	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	checkError(cudaStatus);

	hipFree(d_seq_val_trimmed);
	hipFree(d_seq_trimmed);

	//DB
	/*h_target_index.resize(h_trimmed_seq_size);
	cudaStatus = hipMemcpy(h_target_index.data(), d_sup_database_size, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
	checkError(cudaStatus);*/
	//DBEND

	d_thrust_ptr = thrust::device_pointer_cast(d_new_row_idx_size);
	thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_trimmed_seq_size + 1, d_thrust_ptr);

	d_thrust_ptr = thrust::device_pointer_cast(d_sup_database_size);
	thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_trimmed_seq_size, d_thrust_ptr);

	cudaStatus = hipMemcpy(&h_new_database_size, d_new_row_idx_size + h_trimmed_seq_size, sizeof(int), hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	//DB
	//std::cout << "new database size " << h_new_database_size << '\n';
	//DBEND

	cudaStatus = hipMalloc((void**)&d_new_database_size, sizeof(int));
	checkError(cudaStatus);
	cudaStatus = hipMemcpy(d_new_database_size, d_new_row_idx_size + h_trimmed_seq_size, sizeof(int), hipMemcpyDeviceToDevice);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_target_index, sizeof(int) * h_new_database_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_target_index, 0, sizeof(int) * h_new_database_size);
	checkError(cudaStatus);

	cudaStatus = hipMalloc((void**)&d_new_row_idx_start, sizeof(int) * h_new_database_size);
	checkError(cudaStatus);
	cudaStatus = hipMemset(d_new_row_idx_start, 0, sizeof(int) * h_new_database_size);
	checkError(cudaStatus);

	hipFree(d_old_sup_seq_idx);

	//trackTime(TimeMeasure::other_GPU);

	//DEBUG
	/*h_seq.resize(h_seq_size * h_single_seq_size);
	cudaStatus = hipMemcpy(h_seq.data(), d_seq, sizeof(char) * h_seq_size * h_single_seq_size, hipMemcpyDeviceToHost);
	checkError(cudaStatus);

	h_target_index.resize(h_trimmed_seq_size);
	cudaStatus = hipMemcpy(h_target_index.data(), d_seq_val_idx, sizeof(int) * h_trimmed_seq_size, hipMemcpyDeviceToHost);
	checkError(cudaStatus);*/
	//DEBUG_END

	//repeat until no frequent sequences have been found
	while (h_seq_val.empty() == false) {

		//determine block and grid size
		if (h_database_size > max_block_size) {
			grid_size = { (unsigned int)(h_database_size + max_block_size - 1) / max_block_size, 1, 1 };
			block_size = { max_block_size, 1, 1 };
		}
		else {
			grid_size = { 1, 1, 1 };
			block_size = { (unsigned int)h_database_size, 1, 1 };
		}
		
		//DEBUG
		/*h_target_index.resize(h_new_database_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_target_index, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		//DEBUGEND

		//search database for new sequences 
		scanDatabase <<<grid_size, block_size >>> (d_database_size, d_sup_database_size, d_append, d_assemble, d_row_idx_start, d_database, d_seq_val, d_new_row_idx_start, d_seq_val_idx, d_last_element, d_new_row_idx_size, d_target_index, d_seq_val_trimmed_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);
		
		hipFree(d_append);
		hipFree(d_assemble);
		hipFree(d_last_element);
		hipFree(d_sup_database_size);
		hipFree(d_seq_val_idx);
		hipFree(d_new_row_idx_size);

		//trackTime(TimeMeasure::scan_db);

		//determine block and grid size
		if (h_new_database_size > max_block_size) {
			grid_size = { (unsigned int)(h_new_database_size + max_block_size - 1) / max_block_size, 1, 1 };
			block_size = { max_block_size, 1, 1 };
		}
		else {
			grid_size = { 1, 1, 1 };
			block_size = { (unsigned int)h_new_database_size, 1, 1 };
		}

		//check which rows didn't contain frequent sequence
		prepareVector2 <<<grid_size, block_size >>> (d_new_row_idx_start, d_target_index, d_seq_val, d_new_database_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		//DEBUG
		/*h_target_index.resize(h_new_database_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_target_index, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		//DEBUGEND

		//trackTime(TimeMeasure::other_GPU);

		//prefix sum
		d_thrust_ptr = thrust::device_pointer_cast(d_target_index);
		thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_new_database_size, d_thrust_ptr);

		//trackTime(TimeMeasure::prefix_sum);

		cudaStatus = hipMemcpy(&h_database_size, d_target_index + (h_new_database_size - 1), sizeof(int), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		cudaStatus = hipMemcpy(d_database_size, d_target_index + (h_new_database_size - 1), sizeof(int), hipMemcpyDeviceToDevice);
		checkError(cudaStatus);

		//trackTime(TimeMeasure::VRAM_to_RAM);

		hipFree(d_row_idx_start);

		//DEBUG
		//std::cout <<"database size: " << h_database_size << '\n';
		//DEBUGEND

		cudaStatus = hipMalloc((void**)&d_row_idx_start, sizeof(int) * h_database_size); //HUGE MALLOC HERE
		checkError(cudaStatus);

		//create new database using rows that contained frequent sequences
		streamCompaction <<<grid_size, block_size >>> (d_new_row_idx_start, d_target_index, d_row_idx_start, d_new_database_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		hipFree(d_new_row_idx_start);
		hipFree(d_target_index);

		//trackTime(TimeMeasure::stream_compaction);
		//DEBUG
		/*h_seq_val.resize(h_seq_size);
		cudaStatus = hipMemcpy(h_seq_val.data(), d_seq_val, sizeof(int) * h_seq_val.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		h_seq.resize(h_seq_size * h_single_seq_size);
		cudaStatus = hipMemcpy(h_seq.data(), d_seq, sizeof(char) * h_seq.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		//DEBUGEND

		//Free memory on device
		//hipFree(d_seq_val);
		

		cudaStatus = hipMalloc((void**)&d_target_index, sizeof(int) * h_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMemcpy(d_seq_size, &h_seq_size, sizeof(int), hipMemcpyHostToDevice);
		checkError(cudaStatus);

		//DEBUG
		/*std::cout << "h_seq_size: " << h_seq_size << '\n';*/
		//DEBUGEND

		//determine block and grid size
		if (h_seq_size > max_block_size) {
			grid_size = { (unsigned int)(h_seq_size + max_block_size - 1) / max_block_size, 1, 1 };
			block_size = { max_block_size, 1, 1 };
		}
		else {
			grid_size = { 1, 1, 1 };
			block_size = { (unsigned int)h_seq_size, 1, 1 };
		}

		//check which sequences to trim
		prepareSeq <<<grid_size, block_size >>> (d_seq_val, d_target_index, d_seq_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		//Prefix sum with thrust
		d_thrust_ptr = thrust::device_pointer_cast(d_target_index);
		thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_seq_size, d_thrust_ptr);

		//DEBUG
		/*h_target_index.resize(h_seq_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_target_index, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		std::cout << "d_target_index\n";
		for (int i = 0; i < h_target_index.size(); i++) {
			std::cout << h_target_index[i] << '\n';
		}
		std::cout << "d_target_index\n";*/
		//DEBUG_END

		cudaStatus = hipMemcpy(&h_trimmed_seq_size, d_target_index + (h_seq_size - 1), sizeof(int), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		if (h_trimmed_seq_size == 0) { break; }

		cudaStatus = hipMalloc((void**)&d_seq_val_trimmed, sizeof(int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_seq_trimmed, sizeof(char) * h_trimmed_seq_size * h_single_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_old_sup_seq_idx, sizeof(int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		//trim infrequent sequences
		trimSeq <<<grid_size, block_size >>> (d_seq, d_seq_val, d_seq_trimmed, d_seq_val_trimmed, d_target_index, d_sup_seq_idx, d_old_sup_seq_idx, d_single_seq_size, d_seq_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		//DEBUG
		/*h_debug.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_debug.data(), d_old_sup_seq_idx, sizeof(int) * h_debug.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		std::cout << "d_old_sup_seq_idx\n";
		for (int i = 0; i < h_debug.size(); i++) {
			std::cout << h_debug[i] << '\n';
		}
		std::cout << "d_old_sup_seq_idx_end\n";*/
		//DEBUGEND

		//free memory
		hipFree(d_seq_val);
		hipFree(d_sup_seq_idx);
		hipFree(d_seq);

		cudaStatus = hipMalloc((void**)&d_adj_diff, sizeof(int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		//determine block and grid size
		if (h_trimmed_seq_size > max_block_size) {
			grid_size = { (unsigned int)(h_trimmed_seq_size + max_block_size - 1) / max_block_size, 1, 1 };
			block_size = { max_block_size, 1, 1 };
		}
		else {
			grid_size = { 1, 1, 1 };
			block_size = { (unsigned int)h_trimmed_seq_size, 1, 1 };
		}

		//correct values of d_old_sup_seq_idx so that difference between adjecent values is <= 1
		correctOldSupSeqIdx <<<grid_size, block_size >>> (d_old_sup_seq_idx, d_adj_diff);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		//DEBUG
		//h_target_index.resize(h_trimmed_seq_size);
		//cudaStatus = hipMemcpy(h_target_index.data(), d_adj_diff, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		//checkError(cudaStatus);

		//h_debug.resize(h_trimmed_seq_size);
		//cudaStatus = hipMemcpy(h_debug.data(), d_old_sup_seq_idx, sizeof(int) * h_debug.size(), hipMemcpyDeviceToHost);
		//checkError(cudaStatus);
		//
		//for (int i = 0; i < h_target_index.size(); i++) {
		//	std::cout /*<< h_target_index[i]*/ << ' ' << h_debug[i] << '\n';
		//}
		//DEBUGEND

		d_thrust_ptr = thrust::device_pointer_cast(d_adj_diff);
		thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_trimmed_seq_size, d_thrust_ptr);
		

		thrust::device_ptr<int> d_thrust_ptr2 = thrust::device_pointer_cast(d_old_sup_seq_idx);
		thrust::transform(d_thrust_ptr2, d_thrust_ptr2 + h_trimmed_seq_size, d_thrust_ptr, d_thrust_ptr2, thrust::minus<int>());

		//DEBUG
		/*h_target_index.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_adj_diff, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		////DEBUGEND

		////DEBUG
		/*h_debug.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_debug.data(), d_old_sup_seq_idx, sizeof(int) * h_debug.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		////DEBUGEND

		////DEBUG
		//for (int i = 0; i < h_target_index.size(); i++) {
		//	std::cout << h_target_index[i] << ' ' << h_debug[i] << '\n';
		//}
		//DEBUGEND

		hipFree(d_adj_diff);
		//trackTime(TimeMeasure::other_GPU);
	
		//DEBUG
		/*h_target_index.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_old_sup_seq_idx, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		//DEBUGEND

		//copy results to CPU
		cudaStatus = hipMemcpy(&h_seq_start_size, d_old_sup_seq_idx + (h_trimmed_seq_size - 1), sizeof(int), hipMemcpyDeviceToHost);
		checkError(cudaStatus);
		h_seq_start_size++;

		//DEBUG
		//std::cout << "h_seq_start_size: " << h_seq_start_size << '\n';
		//DEBUGEND

		h_seq_val.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_seq_val.data(), d_seq_val_trimmed, sizeof(int) * h_seq_val.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		h_seq.resize(h_trimmed_seq_size * h_single_seq_size);
		cudaStatus = hipMemcpy(h_seq.data(), d_seq_trimmed, sizeof(char) * h_seq.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		//trackTime(TimeMeasure::VRAM_to_RAM);

		output_string.clear();
		for (int i = 0; i < h_seq_val.size(); i++) {
			for (int j = 0; j < h_single_seq_size; j++) {
				output_string += h_seq[i * h_single_seq_size + j];
			}
			output_string += ' ';
			output_string += std::to_string(h_seq_val[i]);
			output_string += "\n";
		}

		/*h_freq_seq.resize(0);
		for (int i = 0; i < h_seq_val.size(); i++) {
			h_freq_seq.push_back({});
			for (int j = 0; j < h_single_seq_size; j++) {
				h_freq_seq[i].push_back(h_seq[i * h_single_seq_size + j]);
			}
		}*/

		//print frequent sequences //DEBUG
		/*for (int i = 0; i < std::size(h_seq_val); i++) {
			std::cout << h_freq_seq[i] << " " << h_seq_val[i] << "\n";
		}*/

		//write found sequences to output file
		output << output_string;
		/*for (int i = 0; i < std::size(h_seq_val); i++) {
			output << h_freq_seq[i] << " " << h_seq_val[i] << "\n";
		}*/

		/*for (int i = 0; i < std::size(h_seq_val); i++) {
			for (int j = 0; j < h_single_seq_size; j++) {
				output << h_seq[i * h_single_seq_size + j];
			}
			output << " " << h_seq_val[i] << "\n";
		}*/

		//trackTime(TimeMeasure::RAM_to_disk);

		//Prepare new sequences
		h_single_seq_size += 2; //single sequence size = 2n + 1, where n = number of algorithm iterations

		cudaStatus = hipMalloc((void**)&d_seq_start, sizeof(int) * (h_seq_start_size + 1));
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_assemble_start, sizeof(int) * h_seq_start_size);
		checkError(cudaStatus);

		cudaStatus = hipMemcpy(d_single_seq_size, &h_single_seq_size, sizeof(int), hipMemcpyHostToDevice);
		checkError(cudaStatus);

		cudaStatus = hipMemcpy(d_seq_val_trimmed_size, &h_trimmed_seq_size, sizeof(int), hipMemcpyHostToDevice);
		checkError(cudaStatus);

		calculateNewSeqPos <<<grid_size, block_size >>> (d_seq_trimmed, d_seq_start, d_assemble_start, d_old_sup_seq_idx, d_single_seq_size, d_seq_val_trimmed_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		//DEBUG
		/*h_target_index.resize(h_trimmed_seq_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_old_sup_seq_idx, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);*/
		//DEBUGEND

		//Copy data to CPU
		h_seq_start.resize(h_seq_start_size + 1);
		cudaStatus = hipMemcpy(h_seq_start.data(), d_seq_start, sizeof(int) * h_seq_start.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		h_assemble_start.resize(h_seq_start_size);
		cudaStatus = hipMemcpy(h_assemble_start.data(), d_assemble_start, sizeof(int) * h_assemble_start.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		//DEBUG
		/*for (int i = 0; i < h_assemble_start.size(); i++) {
			std::cout << h_seq_start[i] << ' ' << h_assemble_start[i] << '\n';
		}
		std::cout << h_seq_start.back() << '\n';*/
		//DEBUGEND

		//calculate how much space needs to be allocated on GPU for new sequences 
		h_seq_size = 0;
		for (int i = 0; i < h_assemble_start.size(); i++) {
			int n_ap = h_assemble_start[i] - h_seq_start[i]; //number of appends in sub database
			int n_as = h_seq_start[i + 1] - h_assemble_start[i];//number of asemblages in sub database
			h_seq_size += n_ap * n_ap + n_ap * (n_ap - 1) / 2 + n_ap * n_as + n_as * (n_as - 1) / 2;
		}

		cudaStatus = hipMalloc((void**)&d_seq, sizeof(char) * h_seq_size * h_single_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_seq_val, sizeof(int) * h_seq_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_seq_val, 0, sizeof(int) * h_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_sup_seq_idx, sizeof(int) * h_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_append, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_append, 0, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_assemble, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_assemble, 0, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_last_element, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_last_element, 0, sizeof(unsigned int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_seq_val_idx, sizeof(int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_sup_database_size, sizeof(int) * h_trimmed_seq_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_new_row_idx_size, sizeof(int) * (h_trimmed_seq_size + 1));
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_new_row_idx_size, 0, sizeof(int));
		checkError(cudaStatus);

		generateNewSeq <<<grid_size, block_size >>> (d_seq, d_seq_trimmed, d_seq_val_trimmed, d_sup_seq_idx, d_old_sup_seq_idx, d_single_seq_size, d_append,
			d_assemble, d_last_element, d_seq_start, d_assemble_start, d_seq_val_idx, d_sup_database_size, d_new_row_idx_size, d_seq_val_trimmed_size);

		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		checkError(cudaStatus);

		hipFree(d_seq_trimmed);
		hipFree(d_seq_start);
		hipFree(d_assemble_start);
		hipFree(d_old_sup_seq_idx);
		hipFree(d_seq_val_trimmed);

		//DEBUG
		/*h_target_index.resize(h_seq_size);
		cudaStatus = hipMemcpy(h_target_index.data(), d_sup_seq_idx, sizeof(int) * h_target_index.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);
		
		h_seq.resize(h_seq_size * h_single_seq_size);
		cudaStatus = hipMemcpy(h_seq.data(), d_seq, sizeof(char) * h_seq.size(), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		for (int i = 0; i < h_target_index.size(); i++) {
			std::cout << h_target_index[i] << '\n';
		}

		std::cout << '\n';
		*/
		//DEBUG_END

		d_thrust_ptr = thrust::device_pointer_cast(d_new_row_idx_size);
		thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_trimmed_seq_size + 1, d_thrust_ptr);

		d_thrust_ptr = thrust::device_pointer_cast(d_sup_database_size);
		thrust::inclusive_scan(d_thrust_ptr, d_thrust_ptr + h_trimmed_seq_size, d_thrust_ptr);

		cudaStatus = hipMemcpy(&h_new_database_size, d_new_row_idx_size + h_trimmed_seq_size, sizeof(int), hipMemcpyDeviceToHost);
		checkError(cudaStatus);

		if (h_new_database_size == 0) { break; }

		//std::cout << "new database size "<< h_new_database_size << '\n';

		cudaStatus = hipMemcpy(d_new_database_size, d_new_row_idx_size + h_trimmed_seq_size, sizeof(int), hipMemcpyDeviceToDevice);
		checkError(cudaStatus);

		//Check available memory
		/*hipMemGetInfo(&h_free_memory, &h_total_memory);
		std::cout << "free: " << h_free_memory << " total: " << h_total_memory << '\n';*/
		
		//h_requied_memory = 2 * h_new_database_size * sizeof(int);
		//h_partition_num = h_requied_memory / h_free_memory + 1;//+1 - round up
		//h_allocation_size = h_requied_memory / (h_partition_num * 2);

		//h_alloc_success = false;
		//while (h_alloc_success == false) {
		//	cudaStatus = hipMalloc((void**)&d_target_index, h_allocation_size);
		//	if (cudaStatus == hipSuccess) {
		//		cudaStatus = hipMalloc((void**)&d_new_row_idx_start, h_allocation_size);
		//		if (cudaStatus == hipSuccess) {
		//			h_alloc_success = true;
		//		}
		//		else{
		//			h_allocation_size -= 100 * 1024 * 1024; //reduce allocation size by 100 MB until succesful
		//			hipFree(d_target_index);
		//			if (h_allocation_size < 100 * 1024 * 1024) {
		//				break;
		//			}
		//		}
		//	}
		//}
		//
		//if (h_alloc_success == false) {
		//	std::cout << "Error: not enough memory\n";
		//	break;
		//}

		//h_partition_num = h_requied_memory / (h_allocation_size * 2); // number of partitions

		//std::cout << h_partition_num << '\n';

		cudaStatus = hipMalloc((void**)&d_target_index, sizeof(int) * h_new_database_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_target_index, 0, sizeof(int) * h_new_database_size);
		checkError(cudaStatus);

		cudaStatus = hipMalloc((void**)&d_new_row_idx_start, sizeof(int) * h_new_database_size);
		checkError(cudaStatus);
		cudaStatus = hipMemset(d_new_row_idx_start, 0, sizeof(int) * h_new_database_size);
		checkError(cudaStatus);

		//trackTime(TimeMeasure::other_GPU);

		//Check available memory
		/*hipMemGetInfo(&h_free_memory, &h_total_memory);
		std::cout << "free: " << h_free_memory << " total: " << h_total_memory << '\n';*/
	}

	hipFree(d_single_seq_size);
	hipFree(d_seq_val_trimmed_size);
	hipFree(d_new_database_size);
	hipFree(d_database);
	hipFree(d_database_size);

	//trackTime(TimeMeasure::write_results);
	// stop timer
	auto stop = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
	std::cout << "Time taken by function: " << duration.count() << " microseconds\n";
}